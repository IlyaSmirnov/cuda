#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <algorithm>
#include <iostream>

using namespace std;

__global__ void kernel2(double *xtmp, double *x)
{
	const int n = 12;
	const int p = 3;
	const int m = n / p;
	double a[n][n + 1] = { { 10, 4, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1 },
	{ 1, 10, 9, 0, 0, 0, 0, 0, 0, 0, 0, 0, 3 },
	{ 0, 3, 10, 8, 0, 0, 0, 0, 0, 0, 0, 0, 5 },
	{ 0, 0, 2, 10, 8, 0, 0, 0, 0, 0, 0, 0, 1 },
	{ 0, 0, 0, 2, 10, 1, 0, 0, 0, 0, 0, 0, 6 },
	{ 0, 0, 0, 0, 2, 10, 7, 0, 0, 0, 0, 0, 1 },
	{ 0, 0, 0, 0, 0, 4, 10, 6, 0, 0, 0, 0, 1 },
	{ 0, 0, 0, 0, 0, 0, 4, 10, 1, 0, 0, 0, 3 },
	{ 0, 0, 0, 0, 0, 0, 0, 7, 10, 2, 0, 0, 5 },
	{ 0, 0, 0, 0, 0, 0, 0, 0, 7, 10, 1, 0, 1 },
	{ 0, 0, 0, 0, 0, 0, 0, 0, 0, 7, 10, 1, 6 },
	{ 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 9, 10, 1 } };

	int tid = blockIdx.x;

	x[(tid + 1) * m - 1] = xtmp[tid];

	if (tid == 0)
	for (int j = (tid + 1) * m - 2; j >= tid * m; j--)
			x[j] = (a[j][n] - a[j][m - 1] * x[m - 1]) / a[j][j];
	else
	{
		for (int j = (tid + 1) * m - 2; j >= tid * m; j--)
			x[j] = (a[j][n] - a[j][tid * m - 1] * xtmp[tid - 1] - a[j][(tid + 1) * m - 1] * xtmp[tid]) / a[j][j];
	}
}

__global__ void kernel1(double *xtmp)
{
	const int n = 12;
	const int p = 3;
	const int m = n / p;
	double a[n][n + 1] = { { 10, 4, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 1 },
	{ 1, 10, 9, 0, 0, 0, 0, 0, 0, 0, 0, 0, 3 },
	{ 0, 3, 10, 8, 0, 0, 0, 0, 0, 0, 0, 0, 5 },
	{ 0, 0, 2, 10, 8, 0, 0, 0, 0, 0, 0, 0, 1 },
	{ 0, 0, 0, 2, 10, 1, 0, 0, 0, 0, 0, 0, 6 },
	{ 0, 0, 0, 0, 2, 10, 7, 0, 0, 0, 0, 0, 1 },
	{ 0, 0, 0, 0, 0, 4, 10, 6, 0, 0, 0, 0, 1 },
	{ 0, 0, 0, 0, 0, 0, 4, 10, 1, 0, 0, 0, 3 },
	{ 0, 0, 0, 0, 0, 0, 0, 7, 10, 2, 0, 0, 5 },
	{ 0, 0, 0, 0, 0, 0, 0, 0, 7, 10, 1, 0, 1 },
	{ 0, 0, 0, 0, 0, 0, 0, 0, 0, 7, 10, 1, 6 },
	{ 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 9, 10, 1 } };

	int tid = blockIdx.x;

	for (int i = tid * m; i < (tid + 1) * m - 1; i++)
	{
		double tmp = a[i + 1][i] / a[i][i];
		for (int j = 0; j < n + 1; j++)
			a[i + 1][j] -= a[i][j] * tmp;
	}
	
	for (int i = (tid + 1) * m - 3; i > -1 && i >= tid * m - 1; i--)
	{
		double tmp = a[i][i + 1] / a[i + 1][i + 1];
		for (int j = 0; j < n + 1; j++)
			a[i][j] -= a[i + 1][j] * tmp;
	}
	
	double tmp[p][p + 1];

	tmp[0][0] = a[m - 1][m - 1];
	tmp[0][p - 1] = 0;
	for (int i = m; i < n - 1; i++)
	{
		if (a[m - 1][i] != 0)
		{
			tmp[0][1] = a[m - 1][i];
			continue;
		}
	}
	tmp[0][p] = a[m - 1][n];

	tmp[p - 1][0] = 0;
	tmp[p - 1][p] = a[n - 1][n];
	tmp[p - 1][p - 1] = a[n - 1][n - 1];
	for (int i = n - 2; i >= 0; i--)
	{
		if (a[n - 1][i] != 0)
		{
			tmp[p - 1][p - 2] = a[n - 1][i];
			continue;
		}
	}

	for (int i = 1; i < p - 1; i++)
	{
		int k = 0;
		int j = 0;
		while (k < 3)
		{
			if (a[(i + 1) * m - 1][j] != 0)
			{
				tmp[i][k] = a[(i + 1) * m - 1][j];
				k++;
			}
			j++;

		}
		tmp[i][p] = a[(i + 1) * m - 1][n];
	}

	double alpha[p - 1];
	double beta[p - 1];

	alpha[p - 2] = -tmp[p - 1][p - 2] / tmp[p - 1][p - 1];
	beta[p - 2] = tmp[p - 1][p] / tmp[p - 1][p - 1];
	for (int i = p - 3; i >= 0; i--)
	{
		alpha[i] = -tmp[i + 1][i] / (tmp[i + 1][i + 2] * alpha[i + 1] + tmp[i + 1][i + 1]);
		beta[i] = (tmp[i + 1][p] - tmp[i + 1][i + 2] * beta[i + 1]) / (tmp[i + 1][i + 2] * alpha[i + 1] + tmp[i + 1][i + 1]);
	}

	xtmp[0] = (tmp[0][p] - tmp[0][1] * beta[0]) / (tmp[0][1] * alpha[0] + tmp[0][0]);
	for (int i = 1; i < p; i++)
	{
		xtmp[i] = alpha[i - 1] * xtmp[i - 1] + beta[i - 1];
	}

}

int main()
{
	const int n = 12;
	const int p = 3;
	
	double *tmpCuda = NULL;

	hipMalloc((void**) &tmpCuda, p * sizeof(double));

	kernel1 << <p, 1 >> >(tmpCuda);

	double x[n];

	double *xCuda = NULL;

	hipMalloc((void**) &xCuda, n * sizeof(double));

	kernel2 << <p, 1 >> >(tmpCuda, xCuda);

	hipMemcpy(&x, xCuda, n * sizeof(double), hipMemcpyDeviceToHost);

	for (int i = 0; i < n; i++)
		cout << x[i] << endl;

	return 0;
}