#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

using namespace std;

__global__ void xKernel(double *alpha, double *beta, double * x)
{
	int tid = threadIdx.x;

	const int n = 6;
	const int p = (int) (n / 2);
	
	x[p - 1] = (beta[p - 1] + alpha[p - 1] * beta[p + 1]) / (1 - alpha[p + 1] * alpha[p - 1]);

	if (tid == 0)
		for (int i = p - 2; i >= 0; i--)
		{
			x[i] = alpha[i] * x[i + 1] + beta[i];
		}

	if (tid == 1)
		for (int i = p; i < n; i++)
		{
			x[i] = alpha[i + 1] * x[i - 1] + beta[i + 1];
		}
}

__global__ void alphaBetaKernel(double *alpha, double *beta)
{
	int tid = threadIdx.x;

	const int n = 6;
	const int p = (int) (n / 2);
	const double a[n][n + 1] = { { 10, 4, 0, 0, 0, 0, 1 },
	{ 1, 10, 9, 0, 0, 0, 3 },
	{ 0, 3, 10, 8, 0, 0, 5 },
	{ 0, 0, 2, 10, 8, 0, 1 },
	{ 0, 0, 0, 2, 10, 1, 6 },
	{ 0, 0, 0, 0, 2, 10, 1 } };

	alpha[0] = -a[0][1] / a[0][0];
	beta[0] = a[0][n] / a[0][0];
	alpha[n] = -a[n - 1][n - 2] / a[n - 1][n - 1];
	beta[n] = a[n - 1][n] / a[n - 1][n - 1];

	if (tid == 0)
		for (int i = 0; i < p - 1; i++)
		{
			alpha[i + 1] = -a[i + 1][i + 2] / (a[i + 1][i] * alpha[i] + a[i + 1][i + 1]);
			beta[i + 1] = (a[i + 1][n] - a[i + 1][i] * beta[i]) / (a[i + 1][i] * alpha[i] + a[i + 1][i + 1]);
		}
	
	if (tid == 1)
		for (int i = n - 1; i > p - 1; i--)
		{
			alpha[i] = -a[i - 1][i - 2] / (a[i - 1][i] * alpha[i + 1] + a[i - 1][i - 1]);
			beta[i] = (a[i - 1][n] - a[i - 1][i] * beta[i + 1]) / (a[i - 1][i] * alpha[i + 1] + a[i - 1][i - 1]);
		}
}

int main()
{
	const int n = 6;

	double *alphaCuda = NULL;
	double *betaCuda = NULL;

	hipMalloc((void**) &alphaCuda, (n + 1) * sizeof(double));
	hipMalloc((void**) &betaCuda, (n + 1) * sizeof(double));
	
	alphaBetaKernel <<<1, 2 >>>(alphaCuda, betaCuda);
	
	double x[n];
	
	double *xCuda = NULL;

	hipMalloc((void**) &xCuda, n * sizeof(double));

	xKernel << <1, 2 >> >(alphaCuda, betaCuda, xCuda);

	hipMemcpy(&x, xCuda, n * sizeof(double), hipMemcpyDeviceToHost);

	for (int i = 0; i < n; i++)
	{
		cout << x[i] << endl;
	}

	hipFree(alphaCuda);
	hipFree(betaCuda);
	hipFree(xCuda);

    return 0;
}
